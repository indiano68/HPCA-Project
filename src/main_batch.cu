#include "hip/hip_runtime.h"
//Purpose: Main file for benchmarking the mergeSmallBatch_k kernel

#include <iostream>
#include <string>
#include <batch_merge.cuh>
#include <cuda_timing.h>
#include <utils.hpp>

// data type used in the batch merge
using v_type = int;

int main(int argc, char **argv)
{
  printGPUInfo();
  if (argc < 3)
  {
    std::cout << "Usage: " << argv[0] << " <N> " << " <d> " << std::endl;
    abort();
  }

  // Parse command line arguments and ensure they are valid
  unsigned N = std::stoi(argv[1]);
  unsigned d_arg = std::stoi(argv[2]);
  if(d_arg > MAX_BATCH_SIZE || d_arg < 2)
  {
    std::cout << "d must be between 2 and 1024" << std::endl;
    abort();
  }
  unsigned short d = static_cast<unsigned short>(d_arg);

  // Generate random sorted batches to merge
  std::vector<v_type> A_B_vectors(N * d);
  std::vector<unsigned short> A_sizes = build_and_sort_batches(A_B_vectors, N, d);

  std::vector<v_type> vector_out(N * d);
  v_type *v_A_B_gpu;
  unsigned short *v_A_sizes_gpu;

  //timing events
  TIME_EVENT_DEFINE(timing_kernel);TIME_EVENT_CREATE(timing_kernel);
  float timing_kernel_ms = 0.0f;

  hipMalloc(&v_A_B_gpu, vector_sizeof(A_B_vectors));
  hipMalloc(&v_A_sizes_gpu, vector_sizeof(A_sizes));
  hipMemcpy(v_A_B_gpu, A_B_vectors.data(), vector_sizeof(A_B_vectors), hipMemcpyHostToDevice);
  hipMemcpy(v_A_sizes_gpu, A_sizes.data(), vector_sizeof(A_sizes), hipMemcpyHostToDevice);

  //define number of threads and blocks
  unsigned num_threads = min((MAX_BATCH_SIZE / d) * d, N * d);
  unsigned num_blocks = (N * d + num_threads - 1) / num_threads;
  std::cout << "num_threads: " << num_threads << ", num_blocks: " << num_blocks << std::endl;

  //kernel call
  emptyk<<<1, 1>>>();
  TIME_START(timing_kernel);
  mergeSmallBatch_k<<<num_blocks, num_threads>>>(v_A_B_gpu, v_A_sizes_gpu, N, d);
  TIME_STOP_SAVE(timing_kernel, timing_kernel_ms);
  //copy back the result to the host
  hipMemcpy(vector_out.data(), v_A_B_gpu, vector_sizeof(A_B_vectors), hipMemcpyDeviceToHost);

  // Execute CPU version and compare results
  float time_cpu = merge_batch_cpu(A_B_vectors, A_sizes, N, d);
  std::cout << "CPU time: " << time_cpu << " ms" << std::endl;
  std::cout << "Kernel time: " << timing_kernel_ms << " ms" << std::endl;
  std::cout << "Equality: " << (A_B_vectors == vector_out ? "TRUE" : "FALSE") << std::endl;

  hipFree(v_A_B_gpu);
  hipFree(v_A_sizes_gpu);
  TIME_EVENT_DESTROY(timing_kernel);

  return EXIT_SUCCESS;
}