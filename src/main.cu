#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <tools.hpp>
#include <vector>
#include <algorithm>
#include <chrono>
#include <path_merge.cuh>
#include <wrapper.cuh>
#include <thrust_merge.cuh>
#include <cuda_timing.h>

using v_type = int;

int constexpr N_ITER = 1;

__global__ void emptyk()
{
    return;
}

int main(int argc, char **argv)
{

    if (argc < 3)
    {
        std::cout << "Usage: " << argv[0] << " <size-a> " << " <size-b> " << std::endl;
        abort();
    }

    printGPUInfo();
    /*
         Building GPU buffers
    */
    v_type *v_A_gpu, 
           *v_B_gpu, 
           *v_buffer_gpu, 
           *v_out_gpu_0, 
           *v_out_gpu_1,
           *v_out_gpu_2;
    int2   *v_Q_gpu;
    
    /*
        Building vectors to sort 
    */
    std::vector<v_type> vector_A = build_random_vector<v_type>(std::stoi(argv[1]), -1000, 1000);
    std::vector<v_type> vector_B = build_random_vector<v_type>(std::stoi(argv[2]), -1000, 1000);

    /*
        Building buffers for that allow the varius benchmakred kernels 
        to store their output
    */
    int vector_out_size = vector_A.size() + vector_B.size();
    std::vector<v_type> vector_out_0(vector_out_size);
    std::vector<v_type> vector_out_1(vector_out_size);
    std::vector<v_type> vector_out_2(vector_out_size);
    std::vector<v_type> vector_out_3(vector_out_size);
    std::vector<int2> vector_Q;


    float time_0, time_1, time_2, time_3;
    TIME_EVENT_DEFINE(timing_0);TIME_EVENT_CREATE(timing_0);
    TIME_EVENT_DEFINE(timing_1);TIME_EVENT_CREATE(timing_1);
    TIME_EVENT_DEFINE(timing_2);TIME_EVENT_CREATE(timing_2);

    if (vector_A.size() > vector_B.size())
    {
        std::cout << "Required Size A > Size B!" << std::endl;
        abort();
    }

    std::sort(vector_A.begin(), vector_A.end());
    std::sort(vector_B.begin(), vector_B.end());

    if (DEBUG)
    {
        std::cout << "Vector 1: " << std::endl;
        print_vector(vector_A);
        std::cout << "Vector 2: " << std::endl;
        print_vector(vector_B);
    }

    int block_num = (vector_out_size + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    hipMalloc(&v_A_gpu     , vector_sizeof(vector_A));
    hipMalloc(&v_B_gpu     , vector_sizeof(vector_B));
    hipMalloc(&v_out_gpu_0 , vector_sizeof(vector_out_0));
    hipMalloc(&v_out_gpu_1 , vector_sizeof(vector_out_1));
    hipMalloc(&v_out_gpu_2 , vector_sizeof(vector_out_2));
    hipMalloc(&v_Q_gpu     , (block_num) * sizeof(int2));

    vector_Q.resize(block_num);
    hipMemcpy(v_A_gpu, vector_A.data(), vector_sizeof(vector_A), hipMemcpyHostToDevice);
    hipMemcpy(v_B_gpu, vector_B.data(), vector_sizeof(vector_B), hipMemcpyHostToDevice);

    emptyk<<<1, 1>>>();

    /*
    ########################################
        Benchmarking of Erik's Kernel
    ########################################
    */

    TIME_START(timing_0);
    for (int i = 0; i < N_ITER; i++)
    {
        partitioner<<<block_num, THREADS_PER_BLOCK>>>(v_A_gpu, vector_A.size(),
                                                      v_B_gpu, vector_B.size(),
                                                      v_Q_gpu, block_num);

        merge_k_blocked<<<block_num, THREADS_PER_BLOCK>>>(v_A_gpu, vector_A.size(),
                                                          v_B_gpu, vector_B.size(),
                                                          v_out_gpu_0, vector_out_0.size(), v_Q_gpu);
    }
    TIME_STOP_SAVE(timing_0,time_0);

    hipMemcpy(vector_out_0.data(), v_out_gpu_0, vector_sizeof(vector_out_0), hipMemcpyDeviceToHost);
    hipMemcpy(vector_Q.data(), v_Q_gpu, vector_sizeof(vector_Q), hipMemcpyDeviceToHost);

    /* Padding */
    auto remainder = (vector_out_1.size()-1) % THREADS_PER_BLOCK;
    size_t padding = (remainder == 0) ? 0 : THREADS_PER_BLOCK - remainder;
    auto v_buffer = vector_B;
    if (remainder != 0)
    {
        auto biggest_element = std::max(vector_A.back(), vector_B.back());
        v_buffer.resize(v_buffer.size() + padding, biggest_element);
    }
    hipMalloc(&v_buffer_gpu, vector_sizeof(v_buffer));
    hipMemcpy(v_buffer_gpu, v_buffer.data(), vector_sizeof(v_buffer), hipMemcpyHostToDevice);

    /*
    ########################################
        Benchmarking of Triangles Kernel
    ########################################
    */
    TIME_START(timing_1);
    emptyk<<<1, 1>>>();
    for (int i = 0; i < N_ITER; i++)
    {
        merge_k_triangles<<<block_num, THREADS_PER_BLOCK>>>(v_A_gpu, vector_A.size(),
                                                                            v_buffer_gpu, v_buffer.size(),
                                                                            v_out_gpu_1);
    }
    TIME_STOP_SAVE(timing_1,time_1)
    hipMemcpy(vector_out_1.data(), v_out_gpu_1, vector_sizeof(vector_out_1), hipMemcpyDeviceToHost);
    /*
    ########################################
        Benchmarking of Squares Kernel
    ########################################
    */
    TIME_START(timing_2);
    emptyk<<<1, 1>>>();
    for (int i = 0; i < N_ITER; i++)
    {
        partition_k_gpu<<<block_num,1>>>(v_A_gpu,vector_A.size(),
                                                         v_buffer_gpu,v_buffer.size(),
                                                         v_Q_gpu);
        merge_k_gpu_squares<<<block_num,THREADS_PER_BLOCK>>>(v_A_gpu,vector_A.size(),
                                                             v_buffer_gpu,v_buffer.size(),
                                                             v_out_gpu_2,v_Q_gpu);
    }
    TIME_STOP_SAVE(timing_2,time_2);

    hipMemcpy(vector_out_2.data(), v_out_gpu_2, vector_sizeof(vector_out_2), hipMemcpyDeviceToHost);


    time_3 = bench_thrust_merge(vector_A, vector_B, vector_out_3, N_ITER);
    auto merged = mergeSmall_k_cpu(vector_A, vector_B);
    std::cout << "Equality Erik     mergeLarge    : " << (merged == vector_out_0 ? "True " : "False ") << "T " << time_0 / N_ITER << std::endl;
    std::cout << "Equality Triangle mergeLarge    : " << (merged == vector_out_1 ? "True " : "False ") << "T " << time_1 / N_ITER << std::endl;
    std::cout << "Equality Squares  mergeLarge    : " << (merged == vector_out_1 ? "True " : "False ") << "T " << time_2 / N_ITER << std::endl;
    std::cout << "Equality thrust   merge         : " << (merged == vector_out_3 ? "True " : "False ") << "T " << time_3 << std::endl;
    hipFree(v_A_gpu), 
    hipFree(v_B_gpu), 
    hipFree(v_buffer_gpu), 
    hipFree(v_out_gpu_0), 
    hipFree(v_out_gpu_1),
    hipFree(v_out_gpu_2);
    hipFree(v_Q_gpu);
    TIME_EVENT_DESTROY(timing_0);
    TIME_EVENT_DESTROY(timing_1);
    TIME_EVENT_DESTROY(timing_2);

    return EXIT_SUCCESS;
}
