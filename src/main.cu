#include<iostream>
#include<stdio.h>
#include<tools.hpp>
#include<vector>
#include<algorithm>
#include<chrono>
#include<path_merge.cuh>
#include <wrapper.cuh>

using v_type = int;

int main(int argc, char **argv)
{

    if (argc < 3)
    {
        std::cout << "Usage: " << argv[0] << " <size-a> " << " <size-b> " << std::endl;
        abort();
    }
    printGPUInfo();

    std::vector<v_type> vector_1 = build_random_vector<v_type>(std::stoi(argv[1]), -1000, 1000);
    std::vector<v_type> vector_2 = build_random_vector<v_type>(std::stoi(argv[2]), -1000, 1000);
    // std::vector<v_type> vector_1 = {30,50,60,80,110};
    // std::vector<v_type> vector_2 = {10,20,40,70,90,100,120};
    // std::vector<v_type> vector_1 = {1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1};
    // std::vector<v_type> vector_2 = {1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1};
    std::vector<v_type> vector_out0(vector_1.size() + vector_2.size());
    std::vector<v_type> vector_out1(vector_1.size() + vector_2.size());

    v_type *v_1_gpu, *v_2_gpu, *v_out_gpu0, *v_out_gpu1;
    float time0, time1;
    hipEvent_t start, stop;

    std::sort(vector_1.begin(), vector_1.end());
    std::sort(vector_2.begin(), vector_2.end());

    if(DEBUG)
    {
      std::cout << "Vector 1: " << std::endl;
      print_vector(vector_1);
      std::cout << "Vector 2: " << std::endl;
      print_vector(vector_2);
    }

    
    int block_size = (vector_1.size() + vector_2.size()) / 32;

    hipMalloc(&v_1_gpu, vector_sizeof(vector_1));
    hipMalloc(&v_2_gpu, vector_sizeof(vector_2));
    hipMalloc(&v_out_gpu0, vector_sizeof(vector_out0));
    hipMalloc(&v_out_gpu1, vector_sizeof(vector_out0));

    empty_k<<<1, 1>>>();
    hipMemcpy(v_1_gpu, vector_1.data(), vector_sizeof(vector_1), hipMemcpyHostToDevice);
    hipMemcpy(v_2_gpu, vector_2.data(), vector_sizeof(vector_2), hipMemcpyHostToDevice);
    hipFree(0);
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    // for(int i =0; i<100; i++)
    // {
    //     mergeSmall_k2<<<(vector_out0.size() + 1024) / 1024, 1024>>>(v_1_gpu, vector_1.size(),
    //                                                           v_2_gpu, vector_2.size(),
    //                                                           v_out_gpu0, vector_out0.size());
    //     hipDeviceSynchronize();
    // }
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time0, start, stop);

    hipMemcpy(vector_out0.data(), v_out_gpu0, vector_sizeof(vector_out0), hipMemcpyDeviceToHost);
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    // for(int i =0; i<100; i++)
    // {

    //     mergeSmall_k1<<<(vector_out0.size() + 1024) / 1024, 1024>>>(v_1_gpu, vector_1.size(),
    //                                                                 v_2_gpu, vector_2.size(),
    //                                                                 v_out_gpu1, vector_out1.size());
    //     hipDeviceSynchronize();
    // }

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time1, start, stop);
    hipMemcpy(vector_out1.data(), v_out_gpu1, vector_sizeof(vector_out0), hipMemcpyDeviceToHost);
    std::cout << "Computed" << std::endl;
    auto merged = mergeSmall_k_cpu(vector_1, vector_2);

    std::cout << "Equality CPU v1: " << (merged == vector_out1 ? "True " : "False ") << "T " << time1 << std::endl;
    std::cout << "Equality CPU v2: " << (merged == vector_out0 ? "True " : "False ") << "T " << time0 << std::endl;

    auto gpu_merge = call_merge_kernel(vector_1, vector_2);

    auto thrust_merge = merge_arrays_thrust(vector_1, vector_2);
    std::cout << thrust_merge.size() << std::endl;

    if(gpu_merge == merged)
    {
        std::cout<<"TEST PASSED!"<<std::endl;
    }
    else
    {
        std::cout<<"GPU and CPU results are not equal"<<std::endl;
    }

    hipFree(v_1_gpu);
    hipFree(v_2_gpu);
    hipFree(v_out_gpu0);
    hipFree(v_out_gpu1);
    return EXIT_SUCCESS;
}
