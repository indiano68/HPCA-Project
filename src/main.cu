#include "hip/hip_runtime.h"
#include<iostream>
#include<stdio.h>
#include<tools.hpp>
#include<vector>
#include<algorithm>
#include<chrono>
#include <thrust/device_vector.h>
#include <thrust/merge.h>
#include <thrust/execution_policy.h>
#include<path_merge.cuh>
#include <wrapper.cuh>

using v_type = int;

__global__ void emptyk()
{
    return;
}


template <typename T>
std::vector<T> merge_arrays_thrust(const std::vector<T>& A, const std::vector<T>& B)
{
    // Create device vectors from the input std::vectors
    thrust::device_vector<T> d_A = A;
    thrust::device_vector<T> d_B = B;

    thrust::device_vector<T> result(A.size() + B.size());

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    thrust::merge(thrust::device,
                  d_A.begin(), d_A.end(),
                  d_B.begin(), d_B.end(),
                  result.begin());

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    std::cout << "CUDA Thrust merge time: " << milliseconds << "ms" << std::endl;

    // Convert the result back to std::vector and return
    return std::vector<T>(result.begin(), result.end());
}

int main(int argc, char **argv)
{

    if (argc < 3)
    {
        std::cout << "Usage: " << argv[0] << " <size-a> " << " <size-b> " << std::endl;
        abort();
    }
    printGPUInfo();

    std::vector<v_type> vector_1 = build_random_vector<v_type>(std::stoi(argv[1]), -1000, 1000);
    std::vector<v_type> vector_2 = build_random_vector<v_type>(std::stoi(argv[2]), -1000, 1000);
    std::vector<v_type> vector_out0(vector_1.size() + vector_2.size());
    std::vector<v_type> vector_out1(vector_1.size() + vector_2.size());

    v_type *v_1_gpu, *v_2_gpu, *v_out_gpu0, *v_out_gpu1;
    float time0, time1;
    hipEvent_t start, stop;

    std::sort(vector_1.begin(), vector_1.end());
    std::sort(vector_2.begin(), vector_2.end());
    
    int block_size = (vector_1.size() + vector_2.size()) / 32;

    hipMalloc(&v_1_gpu, vector_sizeof(vector_1));
    hipMalloc(&v_2_gpu, vector_sizeof(vector_2));
    hipMalloc(&v_out_gpu0, vector_sizeof(vector_out0));
    hipMalloc(&v_out_gpu1, vector_sizeof(vector_out0));

    emptyk<<<1, 1>>>();
    hipMemcpy(v_1_gpu, vector_1.data(), vector_sizeof(vector_1), hipMemcpyHostToDevice);
    hipMemcpy(v_2_gpu, vector_2.data(), vector_sizeof(vector_2), hipMemcpyHostToDevice);
    hipFree(0);
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    // for(int i =0; i<100; i++)
    {
        mergeSmall_k2<<<(vector_out0.size() + 1024) / 1024, 1024>>>(v_1_gpu, vector_1.size(),
                                                              v_2_gpu, vector_2.size(),
                                                              v_out_gpu0, vector_out0.size());
        hipDeviceSynchronize();
    }
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time0, start, stop);

    hipMemcpy(vector_out0.data(), v_out_gpu0, vector_sizeof(vector_out0), hipMemcpyDeviceToHost);
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    // for(int i =0; i<100; i++)
    {

        mergeSmall_k1<<<(vector_out0.size() + 1024) / 1024, 1024>>>(v_1_gpu, vector_1.size(),
                                                                    v_2_gpu, vector_2.size(),
                                                                    v_out_gpu1, vector_out1.size());
        hipDeviceSynchronize();
    }

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time1, start, stop);
    hipMemcpy(vector_out1.data(), v_out_gpu1, vector_sizeof(vector_out0), hipMemcpyDeviceToHost);
    std::cout << "Computed" << std::endl;
    auto merged = mergeSmall_k_cpu(vector_1, vector_2);

    std::cout << "Equality CPU v1: " << (merged == vector_out1 ? "True " : "False ") << "T " << time1 << std::endl;
    std::cout << "Equality CPU v2: " << (merged == vector_out0 ? "True " : "False ") << "T " << time0 << std::endl;

    auto gpu_merge = call_merge_kernel(vector_1, vector_2);
    auto thrust_merge = merge_arrays_thrust(vector_1, vector_2);
    std::cout << thrust_merge.size() << std::endl;
    if(gpu_merge == merged)
    {
        std::cout<<"TEST PASSED!"<<std::endl;
    }
    else
    {
        std::cout<<"GPU and CPU results are not equal"<<std::endl;
    }

    hipFree(v_1_gpu);
    hipFree(v_2_gpu);
    hipFree(v_out_gpu0);
    hipFree(v_out_gpu1);
    return EXIT_SUCCESS;
}
